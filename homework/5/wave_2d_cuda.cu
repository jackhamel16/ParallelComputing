#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "png_util.h"
#define min(X,Y) ((X) < (Y) ? (X) : (Y))
#define max(X,Y) ((X) > (Y) ? (X) : (Y))
#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}

#define BLOCKDIM 1024

__global__ void array_setup(int nx, int ny, double dx, double dy,
                       double* d_z, double* d_v, double* d_a) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < nx * ny) {
        int row = i / nx;
        int col = i % nx;
	double x = (double)col*dx;
	double y = (double)row*dy;
        d_z[i] = exp(-(sqrt((x-5.0)*(x-5.0)+(y-5.0)*(y-5.0))));
        d_v[i] = 0.0;
	d_a[i] = 0.0;
    }
}

__global__ void accel_update(int nx, int ny, double dx2inv, double dy2inv,
                             double* d_z, double* d_a) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;    
    double ax, ay;
    int r = i / nx;
    int c = i % nx;
    if(i < nx*ny) {
        if(r<ny-1 && r>0 && c<nx-1 && c>0){
            ax = (d_z[i+nx]+d_z[i-nx]-2.0*d_z[i])*dx2inv;
            ay = (d_z[i+1]+d_z[i-1]-2.0*d_z[i])*dy2inv;
            d_a[i] = (ax+ay)/2;
        }
        else
            d_a[i] = 0.0;
    }
}

__global__ void pos_update(int nx, int ny, double dt, double* d_z,
                           double* d_v, double* d_a) {    
    int i = blockDim.x * blockIdx.x + threadIdx.x;    
    int r = i / nx;
    int c = i % nx;
    if(r<ny-1 && r>0 && c<nx-1 && c>0){
        d_v[i] = d_v[i] + dt*d_a[i];
        d_z[i] = d_z[i] + dt*d_v[i];
    }
}
    
int main(int argc, char ** argv) {
    int nx = 500;
    int ny = 500;
    int N = nx * ny;
    int nt = 10000; 
    int frame=0;
    //int nt = 1000000;
    int i,it;
    double dx,dy,dt;
    double max,min;
    double tmax;
    double dx2inv, dy2inv;
    char filename[sizeof "./images/cuda00000.png"];
    double *d_z, *d_v, *d_a, *d_output;

    image_size_t sz; 
    sz.width=nx;
    sz.height=ny;
    int mem_size = nx * ny * sizeof(double);
 
    //make mesh
    double * h_z = (double *) malloc(mem_size);
    //Velocity
    double * h_v = (double *) malloc(mem_size);
    //Accelleration
    double * h_a = (double *) malloc(mem_size);
    //output image
    unsigned char * o_img = (unsigned char *) malloc(sz.width*sz.height*sizeof(unsigned char));

    max=10.0;
    min=0.0;
    dx = (max-min)/(double)(nx-1);
    dy = (max-min)/(double)(ny-1);
    
    tmax=20.0;
    dt= (tmax-0.0)/(double)(nt-1);

    CUDA_CALL(hipMalloc((void **)&d_z, mem_size));
    CUDA_CALL(hipMalloc((void **)&d_v, mem_size));
    CUDA_CALL(hipMalloc((void **)&d_a, mem_size));
    CUDA_CALL(hipMalloc((void **)&d_output, sz.width*sz.height*sizeof(unsigned char)));

    int block_size = BLOCKDIM;
    int num_blocks = N / block_size + 1;
    dim3 dimBlock(block_size, 1, 1);
    dim3 dimGrid(num_blocks, 1, 1);

    array_setup<<<dimGrid, dimBlock>>>(nx, ny, dx, dy, d_z, d_v, d_a);

    dx2inv=1.0/(dx*dx);
    dy2inv=1.0/(dy*dy);
    for(it=0;it<nt-1;it++) {
        accel_update<<<dimGrid, dimBlock>>>(nx, ny, dx2inv, dy2inv, d_z, d_a);
        pos_update<<<dimGrid, dimBlock>>>(nx, ny, dt, d_z, d_v, d_a);    
	if (it % 100 ==0)
	{
            CUDA_CALL(hipMemcpy(h_z, d_z, mem_size, hipMemcpyDeviceToHost));
            double mx,mn;
            mx = -999999;
            mn = 999999;
            for(i=0; i<N; ++i) {
                mx = max(mx, h_z[i]);
                mn = min(mn, h_z[i]);
            }
            for(i=0; i<N; ++i) {
                o_img[i] = (char) round((h_z[i]-mn)/(mx-mn)*255);
            }

    	    sprintf(filename, "./images/cuda%05d.png", frame);
            printf("Writing %s\n",filename);    
            write_png_file(filename,o_img,sz);
            frame+=1;
        }

    }

    CUDA_CALL(hipMemcpy(h_z, d_z, mem_size, hipMemcpyDeviceToHost));
    
    double mx,mn;
    mx = -999999;
    mn = 999999;
    for(i=0; i<N; ++i) {
	 mx = max(mx, h_z[i]);
	 mn = min(mn, h_z[i]);
    }
        
    for(i=0; i<N; ++i) {
        o_img[i] = (char) round((h_z[i]-mn)/(mx-mn)*255);  
    }

    sprintf(filename, "./images/cuda%05d.png", it);
    printf("Writing %s\n",filename);    
    //Write out output image using 1D serial pointer
    write_png_file(filename,o_img,sz);
    return 0;
}
