#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "png_util.h"
#define min(X,Y) ((X) < (Y) ? (X) : (Y))
#define max(X,Y) ((X) > (Y) ? (X) : (Y))
#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}

#define BLOCKDIM 1024

__global__ void array_setup(int nx, int ny, double dx, double dy,
                       double* d_z, double* d_v, double* d_a) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < nx * ny) {
        int row = i / nx;
        int col = i % nx;
	double x = (double)col*dx;
	double y = (double)row*dy;
        d_z[i] = exp(-(sqrt((x-5.0)*(x-5.0)+(y-5.0)*(y-5.0))));
        d_v[i] = 0.0;
	d_a[i] = 0.0;
    }
}

__global__ void accel_update(int nx, int ny, double dx2inv, double dy2inv,
                             double* d_z, double* d_a) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;    
    double ax, ay;
    int r = i / nx;
    int c = i % nx;
    if(r<ny-1 && r>0 && c<nx-1 && c>0){
        ax = (d_z[i+nx]+d_z[i-nx]-2.0*d_z[i])*dx2inv;
        ay = (d_z[i+1]+d_z[i-1]-2.0*d_z[i])*dy2inv;
        d_a[i] = (ax+ay)/2;
    }
//    if(i<(nx*(ny-1)) && i>nx && i%(nx-1)==0 && i%(nx)==0){
//        ax = (d_z[i+nx]+d_z[i-nx]-2.0*d_z[i])*dx2inv;
//        ay = (d_z[i+1]+d_z[i-1]-2.0*d_z[i])*dy2inv;
//        d_a[i] = (ax+ay)/2;
//    }
}

__global__ void pos_update(int nx, int ny, double dt, double* d_z,
                           double* d_v, double* d_a) {    
    int i = blockDim.x * blockIdx.x + threadIdx.x;    
    int r = i / nx;
    int c = i % nx;
    if(r<ny-1 && r>0 && c<nx-1 && c>0){
        d_v[i] = d_v[i] + dt*d_a[i];
        d_z[i] = d_z[i] + dt*d_v[i];
    }
//    if(i<(nx*(ny-1)) && i>nx && i%(nx-1)==0 && i%(nx)==0) {
//        d_v[i] = d_v[i] + dt*d_a[i];
//        d_z[i] = d_z[i] + dt*d_v[i];
//    }
}

int main(int argc, char ** argv) {
    int nx = 500;
    int ny = 500;
    int N = nx * ny;
    int nt = 10000; 
    int frame=0;
    //int nt = 1000000;
    int r,c,it;
    double dx,dy,dt;
    double max,min;
    double tmax;
    double dx2inv, dy2inv;
    char filename[sizeof "./images/cuda00000.png"];
    double *d_z, *d_v, *d_a;

    image_size_t sz; 
    sz.width=nx;
    sz.height=ny;
    int mem_size = nx * ny * sizeof(double);
 
    //make mesh
    //double * h_z = (double *) malloc(mem_size);
    double * h_z = (double *) malloc(mem_size);
    double * h_z2 = (double *) malloc(mem_size);
    double ** z = (double **) malloc(ny * sizeof(double*));
    for (r=0; r<ny; r++)
    	z[r] = &h_z2[r*nx];

    //Velocity
    double * h_v = (double *) malloc(mem_size);
    double * h_v2 = (double *) malloc(mem_size);
    double ** v = (double **) malloc(ny * sizeof(double*));
    for (r=0; r<ny; r++)
        v[r] = &h_v2[r*nx];
    
    //Accelleration
    double * h_a = (double *) malloc(mem_size);
    double * h_a2 = (double *) malloc(mem_size);
    double ** a = (double **) malloc(ny * sizeof(double*));
    for (r=0; r<ny; r++)
        a[r] = &h_a2[r*nx];

    //output image
    unsigned char * o_img = (unsigned char *) malloc(sz.width*sz.height*sizeof(unsigned char));
    unsigned char **output = (unsigned char **) malloc(sz.height * sizeof(unsigned char*));
    for (int r=0; r<sz.height; r++)
        output[r] = &o_img[r*sz.width];

    max=10.0;
    min=0.0;
    dx = (max-min)/(double)(nx-1);
    dy = (max-min)/(double)(ny-1);
    
    tmax=20.0;
    dt= (tmax-0.0)/(double)(nt-1);

    CUDA_CALL(hipMalloc((void **)&d_z, mem_size));
    CUDA_CALL(hipMalloc((void **)&d_v, mem_size));
    CUDA_CALL(hipMalloc((void **)&d_a, mem_size));

    CUDA_CALL(hipMemcpy(d_z, h_z, mem_size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_v, h_v, mem_size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_a, h_a, mem_size, hipMemcpyHostToDevice));

    int block_size=BLOCKDIM;
    int num_blocks = nx / block_size; // nx-2?? also won't this be too low since int division?
    dim3 dimBlock(block_size, 1, 1);
    dim3 dimGrid(num_blocks, 1, 1);
 //   array_setup<<<dimGrid, dimBlock>>>(nx, ny, dx, dy, d_z, d_v, d_a);

    int i;
    double x,y; 
    for (r=0;r<ny;r++)  {
    	for (c=0;c<nx;c++)  {
                i = r * nx + c;
		x = min+(double)c*dx;
		y = min+(double)r*dy;
        	z[r][c] = exp(-(sqrt((x-5.0)*(x-5.0)+(y-5.0)*(y-5.0))));
        	h_z[i] = exp(-(sqrt((x-5.0)*(x-5.0)+(y-5.0)*(y-5.0))));
        	v[r][c] = 0.0;
        	h_v[i] = 0.0;
	        a[r][c] = 0.0;
        	h_a[i] = 0.0;
                //printf("%f \n", z[r][c]); 
    	}
    }
 
    CUDA_CALL(hipMemcpy(d_z, h_z, mem_size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_v, h_v, mem_size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_a, h_a, mem_size, hipMemcpyHostToDevice));
    
//  CUDA_CALL(hipMemcpy(h_z, d_z, mem_size, hipMemcpyDeviceToHost));
//  CUDA_CALL(hipMemcpy(h_v, d_v, mem_size, hipMemcpyDeviceToHost));
//  CUDA_CALL(hipMemcpy(h_a, d_a, mem_size, hipMemcpyDeviceToHost));

//  for(int r=0; r<ny; ++r)
//      for(int c=0; c<nx; ++c) {
//          i = r * nx + c;
//          printf("%f %f \n", h_z[i], z[r][c]);
//      }
    dx2inv=1.0/(dx*dx);
    dy2inv=1.0/(dy*dy);

    for(it=0;it<nt-1;it++) {
	//printf("%d\n",it);
        for (r=1;r<ny-1;r++)  
    	    for (c=1;c<nx-1;c++)  {
		double ax = (z[r+1][c]+z[r-1][c]-2.0*z[r][c])*dx2inv;
		double ay = (z[r][c+1]+z[r][c-1]-2.0*z[r][c])*dy2inv;
		a[r][c] = (ax+ay)/2;
	    }
        accel_update<<<dimGrid, dimBlock>>>(nx, ny, dx2inv, dy2inv, d_z, d_a);
        for (r=1;r<ny-1;r++)  
    	    for (c=1;c<nx-1;c++)  {
               v[r][c] = v[r][c] + dt*a[r][c];
               z[r][c] = z[r][c] + dt*v[r][c];
            }
        pos_update<<<dimGrid, dimBlock>>>(nx, ny, dt, d_z, d_v, d_a);    

	if (it % 100 ==0)
	{
    	    double mx,mn;
    	    mx = -999999;
            mn = 999999;
            for(r=0;r<ny;r++)
                for(c=0;c<nx;c++){
           	    mx = max(mx, z[r][c]);
           	    mn = min(mn, z[r][c]);
        	}
    	    for(r=0;r<ny;r++)
                for(c=0;c<nx;c++)
                    output[r][c] = (unsigned char) round((z[r][c]-mn)/(mx-mn)*255);

    	    sprintf(filename, "./images/cuda%05d.png", frame);
            printf("Writing %s\n",filename);    
    	    write_png_file(filename,o_img,sz);
	    frame+=1;
        }

    }

    CUDA_CALL(hipMemcpy(h_z, d_z, mem_size, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_v, d_v, mem_size, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_a, d_a, mem_size, hipMemcpyDeviceToHost));

    for(int r=0; r<ny; ++r)
        for(int c=0; c<nx; ++c) {
            i = r * nx + c;
            printf("%f %f \n", h_z[i], z[r][c]);
        }
    
    double mx,mn;
    mx = -999999;
    mn = 999999;
    for(r=0;r<ny;r++)
        for(c=0;c<nx;c++){
	   mx = max(mx, z[r][c]);
	   mn = min(mn, z[r][c]);
        }

    printf("%f, %f\n", mn,mx);

    for(r=0;r<ny;r++)
        for(c=0;c<nx;c++){  
	   output[r][c] = (char) round((z[r][c]-mn)/(mx-mn)*255);  
	}

    sprintf(filename, "./images/file%05d.png", it);
    printf("Writing %s\n",filename);    
    //Write out output image using 1D serial pointer
    write_png_file(filename,o_img,sz);
    return 0;
}
